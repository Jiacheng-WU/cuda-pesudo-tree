#include "hip/hip_runtime.h"
#include "test.h"

__global__ void kernel(int* data, int* result) {
    // Kernel code here
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    result[idx] = data[idx] * 2; // Example operation
}

int test() {
    const int arraySize = 5;
    int data[arraySize] = {1, 2, 3, 4, 5};
    int result[arraySize] = {0};

    int *d_data, *d_result;
    hipMalloc((void**)&d_data, arraySize * sizeof(int));
    hipMalloc((void**)&d_result, arraySize * sizeof(int));

    hipMemcpy(d_data, data, arraySize * sizeof(int), hipMemcpyHostToDevice);

    kernel<<<1, arraySize>>>(d_data, d_result);

    hipMemcpy(result, d_result, arraySize * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_result);

    for (int i = 0; i < arraySize; i++) {
        printf("result[%d] = %d\n", i, result[i]);
    }

    return 0;
}

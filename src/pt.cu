#include "hip/hip_runtime.h"
#include "pt.hpp"

#include <cstdint>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <hipcub/hipcub.hpp>

#include <cuda/std/span>

#include <cutlass/cutlass.h>

namespace cg = cooperative_groups;

namespace pt::cuda {

// \sum A_ij * B_ik * C_il
// where A, B, C are (N, N) matrices
// and the output is stored in

constinit const int NUM_THREADS_IN_BLOCK = 1024;

__device__ int64_t sum_over_2nd_dim(const int* Mat, const int N,
                                    const int part_size, int i) {

    int pos_x = i;
    int thread_index_i = threadIdx.x;
    int Mat_i_sum_over_part_j = 0;

    // TODO: need to consider the Memory coalescing
    // change to loop stride mode perhaps at least for warp level
    for (int part_y = 0; part_y < part_size; ++part_y) {
        int pos_y = thread_index_i * part_size + part_y;
        int Mat_ij = Mat[pos_x * N + pos_y];
        Mat_i_sum_over_part_j += Mat_ij;
    }
    cg::this_thread_block().sync();

    using BlockReduce = hipcub::BlockReduce<int, NUM_THREADS_IN_BLOCK>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int Mat_i_sum = BlockReduce(temp_storage).Sum(Mat_i_sum_over_part_j);

    return Mat_i_sum;
}

// Assume N is multiple of NUM_THREADS_IN_BLOCK
__global__ void naive_star_kernel(const int* A, const int* B, const int* C,
                                  const int N, uint64_t* g_output) {
    int block_index_i = blockIdx.x;
    int thread_index_i = threadIdx.x;
    int block_size = blockDim.x;
    [[maybe_unused]] int grid_size = gridDim.x;
    assert(grid_size == block_size);
    assert(block_size == NUM_THREADS_IN_BLOCK);
    assert(N % block_size == 0);

    int part_size = N / block_size;

    uint64_t local_part_output = 0;

    auto block = cg::this_thread_block();

    for (int part_x = 0; part_x < part_size; ++part_x) {
        // load A_ij
        int pos_x = block_index_i * part_size + part_x;

        // This might be inefficient since we sync in each iteration
        // Perhaps we could store the intermediate results in shared memory
        // and do the reduction at the end for each row at same time
        // TODO: Perhaps let each warp do the reduction for one row
        int64_t A_i_sum = sum_over_2nd_dim(A, N, part_size, pos_x);
        int64_t B_i_sum = sum_over_2nd_dim(B, N, part_size, pos_x);
        int64_t C_i_sum = sum_over_2nd_dim(C, N, part_size, pos_x);

        if (thread_index_i == 0) {
            local_part_output += A_i_sum * B_i_sum * C_i_sum;
        }
        block.sync();
    }

    if (thread_index_i == 0) {
        static_assert(sizeof(uint64_t) == sizeof(unsigned long long int));
        atomicAdd(reinterpret_cast<unsigned long long int*>(g_output),
                  static_cast<unsigned long long int>(local_part_output));
    }
}

// A functor to generate random numbers.
// The operator() is called for each element index 'i'.
struct RandomGenerator {
    unsigned long long seed;

    // Constructor captures the seed
    RandomGenerator(unsigned long long s) : seed(s) {}

    // The __host__ __device__ specifiers allow this functor
    // to be created on the host and run on the device.
    __host__ __device__ int operator()(unsigned int i) {
        // Create a new engine for each thread, but seed it uniquely
        // based on the global seed and the thread's index.
        thrust::default_random_engine rng(seed + i);

        // Define the distribution
        thrust::uniform_int_distribution<int> dist(0, 10);

        // Discard the first value to increase randomness, as the first
        // value from a simple LCG can sometimes be weak.
        rng.discard(1);

        // Return a random number from the distribution
        return dist(rng);
    }
};

int64_t cuda_pt_naive(const int N, const unsigned long long seed) {
    assert(N % NUM_THREADS_IN_BLOCK == 0);
    const int num_elements = N * N;

    // 1. Create a Thrust device_vector.
    thrust::device_vector<int> A_matrix(num_elements);
    thrust::device_vector<int> B_matrix(num_elements);
    thrust::device_vector<int> C_matrix(num_elements);

    // 2. Use thrust::tabulate to fill the vector.
    // It calls an instance of our RandomGenerator for each index from 0 to
    // num_elements-1.
    thrust::tabulate(A_matrix.begin(), A_matrix.end(), RandomGenerator(seed));
    thrust::tabulate(B_matrix.begin(), B_matrix.end(), RandomGenerator(seed));
    thrust::tabulate(C_matrix.begin(), C_matrix.end(), RandomGenerator(seed));

    int* raw_ptr_A = thrust::raw_pointer_cast(A_matrix.data());
    int* raw_ptr_B = thrust::raw_pointer_cast(B_matrix.data());
    int* raw_ptr_C = thrust::raw_pointer_cast(C_matrix.data());

    thrust::device_vector<uint64_t> d_output(1, 0);
    uint64_t* raw_ptr_output = thrust::raw_pointer_cast(d_output.data());
    hipDeviceSynchronize();

    naive_star_kernel<<<NUM_THREADS_IN_BLOCK, NUM_THREADS_IN_BLOCK>>>(
        raw_ptr_A, raw_ptr_B, raw_ptr_C, N, raw_ptr_output);
    hipDeviceSynchronize();
    thrust::host_vector<uint64_t> h_output = d_output;

    return h_output[0];
}

int thrust_test() {
    thrust::host_vector<int> data(100);
    thrust::sequence(data.begin(), data.end());
    thrust::host_vector<int> result(data.size());
    thrust::device_vector<int> d_data(data.begin(), data.end());
    thrust::device_vector<int> d_result(result.size());

    thrust::transform(d_data.begin(), d_data.end(), d_result.begin(),
                      [] __device__(int x) { return x * 2; });

    thrust::copy(d_result.begin(), d_result.end(), result.begin());
    return 0;
}

} // namespace pt::cuda
